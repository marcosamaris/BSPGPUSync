#include "hip/hip_runtime.h"
//#include <prof.cu>
// Includes
#include <stdio.h>
#include <float.h>
#include <assert.h>     
#include <hip/hip_runtime_api.h>
//#include <cutil_inline.h>

#define DECIMAL_DIG 12

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}


// Variables
float* h_A;
float* h_B;
float* d_A;

// Functions
void RandomInit(float*, int);

// Device code
__global__ void transFloat(float* A, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        i++;
}

// Host code
int main(int argc, char** argv)
{

    if (argc != 3 ) {
		fprintf(stderr, "Syntax: %s <Vector size>  <device>\n", argv[0]);
    		return EXIT_FAILURE;
	}
    hipProfilerStart();
    
    int N = atoi(argv[1]);    
    size_t size = N * sizeof(float);
    
    int devId = atoi(argv[2]);
    checkCuda( hipSetDevice(devId) );
    hipDeviceReset();

    hipDeviceProp_t prop;
    checkCuda( hipGetDeviceProperties(&prop, devId) );
    printf("Device: %s\n", prop.name);

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);

    // Initialize input vectors
    RandomInit(h_A, N);

    // Allocate vectors in device memory
    checkCuda(hipMalloc((void**)&d_A, size * sizeof(float))) ;

    // Copy vectors from host memory to device memory
    checkCuda(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice)) ;

    for(int i; i<10; i++) printf("%f, ", h_A[i]);
    printf("\n");

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
//	GpuProfiling::prepareProfiling( blocksPerGrid, threadsPerBlock );
    transFloat<<<blocksPerGrid, threadsPerBlock>>>(d_A, N);
//	GpuProfiling::addResults("VecAdd");
//    cutilCheckMsg("kernel launch failure");
#ifdef _DEBUG
    checkCuda( hipDeviceSynchronize() );
#endif

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    checkCuda(hipMemcpy(h_B, d_A, size, hipMemcpyDeviceToHost)) ;

    for(int i; i<10; i++) printf("%f, ", h_B[i]);
    printf("\n");

    free(h_A);
    free(h_B);
    hipFree(d_A);
    hipDeviceReset() ;

    return 0;
    hipProfilerStop();

}



// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = ((rand() / (float)RAND_MAX)*FLT_MAX) + (rand() / (float)RAND_MAX);
}

