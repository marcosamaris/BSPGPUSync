#include "hip/hip_runtime.h"
#include "common.h"

/* Convenience function for checking CUDA runtime API results can be
 * wrapped around any runtime API call. No-op in release builds. */
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

double calculate_elapsed_time(struct timespec start, struct timespec finish)
{
  double elapsed;
  elapsed = (finish.tv_sec - start.tv_sec);
  elapsed += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;
  return elapsed;
}

bool compare_doubles(double n1, double n2, double epsilon)
{
  return fabs(n1 - n2) < epsilon;
}
