#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/*
  Implementing Breadth first search on CUDA using algorithm given in DAC'10
  paper "An Effective GPU Implementation of Breadth-First Search"

  Copyright (c) 2010 University of Illinois at Urbana-Champaign. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
  OTHERWISE.

  Author: Lijiuan Luo (lluo3@uiuc.edu)
  Revised for Parboil 2 Benchmark Suite by: Geng Daniel Liu (gengliu2@illinois.edu)
*/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <parboil.h>
#include <deque>
#include <iostream>

#include <stdint.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
double wall_time() {
        static bool first_call = true;
        static double start_time;

        struct timeval tv;
        gettimeofday(&tv,0);
        double now = tv.tv_sec + 1e-6*tv.tv_usec;

        if (first_call) {
            first_call = false;
            start_time = now;
        }
        return now - start_time;
    }

/// \brief Wrappers around platform dependent timers and performance info

    /// Returns the wall time in seconds relative to arbitrary origin
    /// As accurate and lightweight as we can get it, but may not
    /// be any better than the gettime of day system call.

    /// On some machines we have access to a cycle count

    /// Otherwise uses wall_time() in nanoseconds.
    static inline uint64_t cycle_count() {
        uint64_t x;
        unsigned int a,d;
__asm__ volatile("rdtsc" : "=a"(a), "=d"(d));
        x = ((uint64_t)a) | (((uint64_t)d)<<32);
        return x;
    }

    double cpu_frequency() {
        static double freq = -1.0;
        if (freq == -1.0) {
            double used = wall_time();
            uint64_t ins = cycle_count();
            if (ins == 0) return 0;
            while ((cycle_count()-ins) < 100000000);  // 100M cycles at 1GHz = 0.1s
            ins = cycle_count() - ins;
            used = wall_time() - used;
            freq = ins/used;
        }
        return freq;
    }
    /// Returns the cpu time in seconds relative to arbitrary origin

    /// As accurate and lightweight as we can get it, but may not
    /// be any better than the clock system call.
    static inline double cpu_time() {
        static const double rfreq = 1.0/cpu_frequency();
        return cycle_count()*rfreq;
    }


    /// Do nothing and especially do not touch memory
    inline void cpu_relax() {
        asm volatile("rep;nop" : : : "memory");
    }

    /// Sleep or spin for specified no. of microseconds

    /// Wrapper to ensure desired behavior (and what is that one might ask??)
       // usleep(us);


static double tttt, ssss;
#define STARTt_TIMER  tttt=wall_time(); ssss=cpu_time()
#define ENDt_TIMER(msg) tttt=wall_time()-tttt; ssss=cpu_time()-ssss;  printf("timer: %20.20s %8.10fs %8.10fs\n", msg, ssss, tttt)


#define MAX_THREADS_PER_BLOCK 512
#define NUM_SM 14 //the number of Streaming Multiprocessors; 15 for Fermi architecture 30 for G280 at the moment of this document
#define NUM_BIN 8 //the number of duplicated frontiers used in BFS_kernel_multi_blk_inGPU
#define EXP 3 // EXP = log(NUM_BIN), assuming NUM_BIN is still power of 2 in the future architecture
	//using EXP and shifting can speed up division operation 
#define MOD_OP 7 // This variable is also related with NUM_BIN; may change in the future architecture;
	//using MOD_OP and "bitwise and" can speed up mod operation
#define INF 2147483647//2^31-1
int no_of_nodes; //the number of nodes in the graph
int edge_list_size;//the number of edges in the graph
FILE *fp;

typedef int2 Node;
typedef int2 Edge;

#include "kernel.cu"
//Somehow "hipMemset" does not work. So I use hipMemcpy of constant variables for initialization
const int h_top = 1;
const int zero = 0;

void runGPU(int argc, char** argv);
////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    //printf(" device 1 \n");
	no_of_nodes=0;
	edge_list_size=0;
    hipSetDevice(0);
	runGPU(argc,argv);

}
bool  BFS_GPU( Node * d_graph_nodes,Edge * d_graph_edges,
	int * d_color, int * d_cost, int * d_q1, int * d_q2, int * tail, int source, int & global_kt){
	int num_of_blocks; 
	int num_of_threads_per_block;
//	hipFuncSetCacheConfig(reinterpret_cast<const void*>(BFS_in_GPU_kernel), hipFuncCachePreferShared);    
//	hipFuncSetCacheConfig(reinterpret_cast<const void*>(BFS_kernel_multi_blk_inGPU), hipFuncCachePreferShared);    
//	hipFuncSetCacheConfig(reinterpret_cast<const void*>(BFS_kernel), hipFuncCachePreferShared);    

	(hipMemcpy(tail,&h_top,sizeof(int),hipMemcpyHostToDevice));
	(hipMemcpy(&d_cost[source],&zero,sizeof(int),hipMemcpyHostToDevice));

	( hipMemcpy( &d_q1[0], &source, sizeof(int), hipMemcpyHostToDevice) );
	int num_t;//number of threads
	int k=0;//BFS level index
	//hipHostRegister(&num_t,sizeof(int),hipHostRegisterDefault);
	//whether or not to adjust "k", see comment on "BFS_kernel_multi_blk_inGPU" for more details 
	int * switch_kd;
	( hipMalloc( (void**) &switch_kd, sizeof(int)));
	int * num_td;//number of threads
	( hipMalloc( (void**) &num_td, sizeof(int)));

	//whether to stay within a kernel, used in "BFS_kernel_multi_blk_inGPU"
	bool *stay;
	( hipMalloc( (void**) &stay, sizeof(bool)));
	int switch_k;

	//max number of frontier nodes assigned to a block
	int * max_nodes_per_block_d;
	( hipMalloc( (void**) &max_nodes_per_block_d, sizeof(int)));
	#ifdef DIS_COMP
	int max_nodes_per_block;
	#endif
    int *global_kt_d;
	( hipMalloc( (void**) &global_kt_d, sizeof(int)));
	(hipMemcpy(global_kt_d,&global_kt, sizeof(int),hipMemcpyHostToDevice));
	
	do
	{
//STARTt_TIMER;
		( hipMemcpy( &num_t, tail, sizeof(int), hipMemcpyDeviceToHost) );
		(hipMemcpy(tail,&zero,sizeof(int),hipMemcpyHostToDevice));

//ENDt_TIMER("transfer");
	//	printf("BFS_LEVEL=%d elements=%d\n",global_kt,num_t);
		if(num_t == 0){//frontier is empty
			(hipFree(stay));
			(hipFree(switch_kd));
			(hipFree(num_td));
			(hipMemcpy(&global_kt,global_kt_d, sizeof(int),hipMemcpyDeviceToHost));
	//		hipHostUnregister(&num_t);
			return false;
		}

num_of_blocks = 1;
		num_of_threads_per_block = num_t;
		if(num_of_threads_per_block <NUM_BIN)
			num_of_threads_per_block = NUM_BIN;
		if(num_t>MAX_THREADS_PER_BLOCK)
		{
			num_of_blocks = (int)ceil(num_t/(double)MAX_THREADS_PER_BLOCK); 
			num_of_threads_per_block = MAX_THREADS_PER_BLOCK;
		}
		if(num_of_blocks == 1)//will call "BFS_in_GPU_kernel" 
			num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
		if(num_of_blocks >1 && num_of_blocks <= NUM_SM)// will call "BFS_kernel_multi_blk_inGPU"
			num_of_blocks = NUM_SM;

		//assume "num_of_blocks" can not be very large
	        dim3  grid( num_of_blocks, 1, 1);
		dim3  threads( num_of_threads_per_block, 1, 1);
//printf("BFS_LEVEL=%d elements=%d\n",k,num_t);
		if(k%2 == 0){
			if(num_of_blocks == 1){
//STARTt_TIMER;
				BFS_in_GPU_kernel<<< grid, threads >>>(d_q1,d_q2, d_graph_nodes, 
				 d_graph_edges, d_color, d_cost,num_t , tail,GRAY0,k );
//	(hipDeviceSynchronize());
//ENDt_TIMER("kernelll1");
			}
			else if(num_of_blocks <= NUM_SM){
				#ifdef DIS_COMP
				max_nodes_per_block = ceil(float(num_t)/NUM_SM);
				(hipMemcpy(max_nodes_per_block_d,
					&max_nodes_per_block,sizeof(int), hipMemcpyHostToDevice));
				#endif
				(hipMemcpy(num_td,&num_t,sizeof(int),
					hipMemcpyHostToDevice));
//STARTt_TIMER;
				BFS_kernel_multi_blk_inGPU
				<<< grid, threads >>>(d_q1,d_q2, d_graph_nodes, 
				 d_graph_edges, d_color, d_cost, num_td, tail,GRAY0,k,
				switch_kd, max_nodes_per_block_d, global_kt_d);
//	(hipDeviceSynchronize());
//ENDt_TIMER("kernell1");
				(hipMemcpy(&switch_k,switch_kd, sizeof(int),
				hipMemcpyDeviceToHost));
				if(!switch_k){
					k--;
				}
			}
			else{
//STARTt_TIMER;
				BFS_kernel<<< grid, threads >>>(d_q1,d_q2, d_graph_nodes, 
				 d_graph_edges, d_color, d_cost, num_t, tail,GRAY0,k);
//	(hipDeviceSynchronize());
//ENDt_TIMER("kernel1");
			}
		}
		else{
			if(num_of_blocks == 1){
//STARTt_TIMER;
				BFS_in_GPU_kernel<<< grid, threads >>>(d_q2,d_q1, d_graph_nodes, 
				 d_graph_edges, d_color, d_cost, num_t, tail,GRAY1,k);
//	(hipDeviceSynchronize());
//ENDt_TIMER("kernelll2");
			}
			else if(num_of_blocks <= NUM_SM){
				#ifdef DIS_COMP
				max_nodes_per_block = ceil(float(num_t)/NUM_SM);
				(hipMemcpy(max_nodes_per_block_d,
					&max_nodes_per_block,sizeof(int), hipMemcpyHostToDevice));
				#endif
				(hipMemcpy(num_td,&num_t,sizeof(int),
					hipMemcpyHostToDevice));
//STARTt_TIMER;
				BFS_kernel_multi_blk_inGPU
				<<< grid, threads >>>(d_q2,d_q1, d_graph_nodes, 
				 d_graph_edges, d_color, d_cost, num_td, tail,GRAY1,k,
				 switch_kd, max_nodes_per_block_d, global_kt_d);
//	(hipDeviceSynchronize());
//ENDt_TIMER("kernell2");
				(hipMemcpy(&switch_k,switch_kd, sizeof(int),
				hipMemcpyDeviceToHost));
				if(!switch_k){
					k--;
				}
			}
			else{
//STARTt_TIMER;
				BFS_kernel<<< grid, threads >>>(d_q2,d_q1, d_graph_nodes, 
				 d_graph_edges, d_color, d_cost, num_t, tail, GRAY1,k);
//	(hipDeviceSynchronize());
//ENDt_TIMER("kernel2");
			}
		}
			
		// check if kernel execution generated any error
//ENDt_TIMER("kernel");
		//CUT_CHECK_ERROR("Kernel execution failed");
		
		k++;
	}
	while(1);
}
///////////////////////////////
//FUNCTION:only run GPU version 
////////////////////////////////////////////
void runGPU( int argc, char** argv) 
{

    struct pb_Parameters *params;
    struct pb_TimerSet timers;

    pb_InitializeTimerSet(&timers);
    params = pb_ReadParameters(&argc, argv);
    if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] != NULL))
    {
        fprintf(stderr, "Expecting one input filename\n");
        exit(-1);
    }

    pb_SwitchToTimer(&timers, pb_TimerID_IO);
	//printf("Reading File\n");
	//Read in Graph from a file
	fp = fopen(params->inpFiles[0],"r");
	if(!fp)
	{
		printf("Error Reading graph file\n");
		return;
	}
	int source;

	fscanf(fp,"%d",&no_of_nodes);
	// allocate host memory
	Node* h_graph_nodes = (Node*) malloc(sizeof(Node)*no_of_nodes);
	int *color = (int*) malloc(sizeof(int)*no_of_nodes);
	int start, edgeno;   
	// initalize the memory
	for( unsigned int i = 0; i < no_of_nodes; i++) 
	{
		fscanf(fp,"%d %d",&start,&edgeno);
		h_graph_nodes[i].x = start;
		h_graph_nodes[i].y = edgeno;
		color[i]=WHITE;
	}
	//read the source node from the file
	fscanf(fp,"%d",&source);
	fscanf(fp,"%d",&edge_list_size);
	int id,cost;
	Edge* h_graph_edges = (Edge*) malloc(sizeof(Edge)*edge_list_size);
	for(int i=0; i < edge_list_size ; i++)
	{
		fscanf(fp,"%d",&id);
		fscanf(fp,"%d",&cost);
		h_graph_edges[i].x = id;
		h_graph_edges[i].y = cost;
	}
	if(fp)
		fclose(fp);    

//	printf("Read File\n");

	// allocate mem for the result on host side
	int* h_cost = (int*) malloc( sizeof(int)*no_of_nodes);
	for(int i = 0; i < no_of_nodes; i++){
		h_cost[i] = INF;
	}
	h_cost[source] = 0;

    pb_SwitchToTimer(&timers, pb_TimerID_COPY);

	int * temp = NULL;
	( hipMalloc( (void**) &temp, sizeof(int)*no_of_nodes) );
	( hipFree( temp) );
//	unsigned int copy_timer = 0;
//	cutilCheckError(cutCreateTimer(&copy_timer));
//	cutilCheckError(cutStartTimer(copy_timer));

	//Copy the Node list to device memory
	Node* d_graph_nodes;
	( hipMalloc( (void**) &d_graph_nodes, sizeof(Node)*no_of_nodes) );
	( hipMemcpy( d_graph_nodes, h_graph_nodes, sizeof(Node)*no_of_nodes, hipMemcpyHostToDevice) );
	//Copy the Edge List to device Memory
	Edge* d_graph_edges;
	( hipMalloc( (void**) &d_graph_edges, sizeof(Edge)*edge_list_size) );
	( hipMemcpy( d_graph_edges, h_graph_edges, sizeof(Edge)*edge_list_size, hipMemcpyHostToDevice) );


	int* d_color;
	( hipMalloc( (void**) &d_color, sizeof(int)*no_of_nodes) );
	int* d_cost;
	( hipMalloc( (void**) &d_cost, sizeof(int)*no_of_nodes));
	int * d_q1;
	int * d_q2;
	( hipMalloc( (void**) &d_q1, sizeof(int)*no_of_nodes));
	( hipMalloc( (void**) &d_q2, sizeof(int)*no_of_nodes));
	int * tail;
	( hipMalloc( (void**) &tail, sizeof(int)));
	int *front_cost_d;
	( hipMalloc( (void**) &front_cost_d, sizeof(int)));
	( hipMemcpy( d_color, color, sizeof(int)*no_of_nodes, hipMemcpyHostToDevice) );
	( hipMemcpy( d_cost, h_cost, sizeof(int)*no_of_nodes, hipMemcpyHostToDevice) );
	
//	printf("Copied Everything to GPU memory\n");


	//bind the texture memory with global memory
	(hipBindTexture(0,g_graph_node_ref,d_graph_nodes, sizeof(Node)*no_of_nodes));
	(hipBindTexture(0,g_graph_edge_ref,d_graph_edges,sizeof(Edge)*edge_list_size));

	int cur_count = 0;
	
    printf("Starting GPU kernel\n");
	(hipDeviceSynchronize());
    pb_SwitchToTimer(&timers, pb_TimerID_GPU);
//STARTt_TIMER;
	BFS_GPU( d_graph_nodes,d_graph_edges,
		d_color, d_cost, d_q1, d_q2,  tail, source, cur_count);
	(hipDeviceSynchronize());
//ENDt_TIMER("bfs gpu");
    pb_SwitchToTimer(&timers, pb_TimerID_COPY);
    printf("GPU kernel done\n");


	// copy result from device to host
	( hipMemcpy( h_cost, d_cost, sizeof(int)*no_of_nodes, hipMemcpyDeviceToHost) );
	( hipMemcpy( color, d_color, sizeof(int)*no_of_nodes, hipMemcpyDeviceToHost) );
	( hipUnbindTexture(g_graph_node_ref) );
	( hipUnbindTexture(g_graph_edge_ref) );

	(hipFree(d_graph_nodes));
	(hipFree(d_graph_edges));
	(hipFree(d_color));
	(hipFree(d_cost));
	(hipFree(tail));
	(hipFree(front_cost_d));
	//Store the result into a file
    pb_SwitchToTimer(&timers, pb_TimerID_IO);
	FILE *fp = fopen(params->outFile,"w");
    fprintf(fp, "%d\n", no_of_nodes);
	for(int i=0;i<no_of_nodes;i++)
		fprintf(fp,"%d %d\n",i,h_cost[i]);
	fclose(fp);
	//printf("Result stored in %s\n", params->outFile);

	// cleanup memory
	free( h_graph_nodes);
	free( h_graph_edges);
	free( color);
	free( h_cost);
    pb_SwitchToTimer(&timers, pb_TimerID_NONE);
    pb_PrintTimerSet(&timers);
    pb_FreeParameters(params);
}
