#include "hip/hip_runtime.h"
/*
 * bitonic_sort.cu
 *
 */
#include <assert.h> 
#include <math.h>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
//#include <cutil_inline.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}
 
 #define 	MAX_THREADS 	128
 
 int* r_values;
 int* d_values;

 void Init(int* values, int i, int N) {
        srand( time(NULL) );
	printf("\n------------------------------\n");
 
        if (i == 0) {
        // Uniform distribution
                printf("Data set distribution: Uniform\n");
                for (int x = 0; x < N; ++x) {
                        values[x] = rand() % 100;
                        //printf("%d ", values[x]);
                }
        }
        else if (i == 1) {
        // Gaussian distribution
        #define MEAN    100
        #define STD_DEV	5 
                printf("Data set distribution: Gaussian\n");
                float r;
                for (int x = 0; x < N; ++x) {
                        r  = (rand()%3 - 1) + (rand()%3 - 1) + (rand()%3 - 1);
                        values[x] = int( round(r * STD_DEV + MEAN) );
                        //printf("%d ", values[x]);
                }
        }
        else if (i == 2) {
        // Bucket distribution
                printf("Data set distribution: Bucket\n");
                int j = 0;
                for (int x = 0; x < N; ++x, ++j) {
                        if (j / 20 < 1)
                                values[x] = rand() % 20;
                        else if (j / 20 < 2)
                                values[x] = rand() % 20 + 20;
                        else if (j / 20 < 3)
                                values[x] = rand() % 20 + 40;
                        else if (j / 20 < 4)
                                values[x] = rand() % 20 + 60;
                        else if (j / 20 < 5)
                                values[x] = rand() % 20 + 80; 
                        if (j == 100)
                                j = 0;
                        //printf("%d ", values[x]);
                }
        }
        else if (i == 3) {
        // Sorted distribution
                printf("Data set distribution: Sorted\n");
                /*for (int x = 0; x < N; ++x)
                        print("%d ", values[x]);
		*/
 	}
        else if (i == 4) {
        // Zero distribution
                printf("Data set distribution: Zero\n");
                int r = rand() % 100;
                for (int x = 0; x < N; ++x) {
                        values[x] = r;
                        //printf("%d ", values[x]);
                }
        }
	printf("\n");
}
 
 // Kernel function
 __global__ static void Bitonic_Sort(int* values, int j, int k, int N) {
 	const unsigned long int idx = blockDim.x * blockIdx.x + threadIdx.x;

 	if (idx < N) {
 		int ixj = idx^j;
  		if (ixj > idx) {
 			if ((idx&k) == 0 && values[idx] > values[ixj]) {
				//exchange(idx, ixj);
				int tmp = values[idx];
				values[idx] = values[ixj];
				values[ixj] = tmp;
			}
			if ((idx&k) != 0 && values[idx] < values[ixj]) {
				//exchange(idx, ixj);
				int tmp = values[idx];
				values[idx] = values[ixj];
				values[ixj] = tmp;
			}
 		}	
 	}
}
 
 // program main
 int main(int argc, char** argv) {

	if (argc != 4) {
		fprintf(stderr, "Syntax: %s <Vector size Width> <CacheConfL1> <Device>\n", argv[0]);
    		return EXIT_FAILURE;
	}
    hipProfilerStart();

	int N = atoi(argv[1]);
  	int CacheConfL1 = atoi(argv[2]);
    int devId = atoi(argv[3]);
	size_t size = N * sizeof(long int);

	checkCuda( hipSetDevice(devId) );
    hipDeviceReset();
	
	hipDeviceProp_t prop;
	checkCuda( hipGetDeviceProperties(&prop, devId) );
  	printf("Device: %s\n", prop.name);

	
	printf("./bitonic_sort starting with %d numbers...\n", N);
	srand( time(NULL) );

 	
 	// allocate host memory
 	r_values = (int*)malloc(size);
	
	// allocate device memory
	hipMalloc((void**)&d_values, size);
 	
	/* Types of data sets to be sorted:
	 *	1. Normal distribution
	 *	2. Gaussian distribution
	 *	3. Bucket distribution
	 *	4. Sorted Distribution
	 *	5. Zero Distribution
	 */

 	for (int i = 0; i < 1; ++i) {
		// initialize data set
 		Init(r_values, i, N);
 		 
 		// copy data to device
 		hipMemcpy(d_values, r_values, size, hipMemcpyHostToDevice) ;

		printf("Beginning kernel execution...\n");
		
		if (CacheConfL1 == 1){
	        hipFuncSetCacheConfig(reinterpret_cast<const void*>(Bitonic_Sort), hipFuncCachePreferShared);
	    }
	    else if (CacheConfL1 == 2){
    	    hipFuncSetCacheConfig(reinterpret_cast<const void*>(Bitonic_Sort), hipFuncCachePreferEqual);
	    }
	    else if (CacheConfL1 == 3){
	        hipFuncSetCacheConfig(reinterpret_cast<const void*>(Bitonic_Sort), hipFuncCachePreferL1);
	    }
	    else {
	        hipFuncSetCacheConfig(reinterpret_cast<const void*>(Bitonic_Sort), hipFuncCachePreferNone);
	    }
 		
 		hipDeviceSynchronize() ;
		// execute kernel
       		 
		for (int k = 2; k <= N; k <<= 1) {
			for (int j = k >> 1; j > 0; j = j >> 1) {
				if (N < MAX_THREADS)
					Bitonic_Sort <<< 1, N >>> (d_values, j, k, N);
				else
					Bitonic_Sort <<< N / MAX_THREADS, MAX_THREADS >>> (d_values, j, k, N);
			}
		}
        	 		
 
		hipDeviceSynchronize() ;
 
 		// copy data back to host
		hipMemcpy(r_values, d_values, size, hipMemcpyDeviceToHost) ;

		// test
		printf("Assertion started\n");
		for (int x = 0; x < N - 1; x++) 
            assert(r_values[x] <= r_values[x + 1]);
        printf("Assertion Finished");
	}

 	// free memory
 	hipFree(d_values);
 	free(r_values);	
 	
 	hipDeviceReset();
hipProfilerStop();
  
}
