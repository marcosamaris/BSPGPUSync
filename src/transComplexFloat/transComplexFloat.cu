#include <complex>
#include <stdio.h>
#include <cmath>
#include <float.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h>


inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

hipFloatComplex *h_A;
hipFloatComplex *h_B;
hipFloatComplex z;

__global__ void transComplexFloat(hipFloatComplex *array, int N){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        i++;
}

int main(int argc, char** argv ) {
    
    if (argc != 3 ) {
        fprintf(stderr, "Syntax: %s <Vector size>  <device>\n", argv[0]);
            return EXIT_FAILURE;
    }
    hipProfilerStart();
    
    int N = atoi(argv[1]);    
    size_t size = N * sizeof(float);
    
    int devId = atoi(argv[2]);
    checkCuda( hipSetDevice(devId) );
    hipDeviceReset();

    hipDeviceProp_t prop;
    checkCuda( hipGetDeviceProperties(&prop, devId) );
    printf("Device: %s\n", prop.name);

    h_A = new hipFloatComplex[N];
    h_B = new hipFloatComplex[N];

    for(int i = 0; i < N; ++i){
        float Ti = ((rand() / (float)RAND_MAX)*FLT_MAX) + (rand() / (float)RAND_MAX);
        float Tj = ((rand() / (float)RAND_MAX)*FLT_MAX) + (rand() / (float)RAND_MAX);
        z = make_hipFloatComplex(Ti, Tj);
        h_A[i] = make_hipFloatComplex(hipCrealf(z), hipCimagf(z));
        h_B[i] = make_hipFloatComplex(0., 0.);      
    }

  
  //Allocate and copy memory to device
  hipFloatComplex *d_A;
  checkCuda(hipMalloc((void**)&d_A, sizeof(hipFloatComplex)*N));
  checkCuda(hipMemcpy(d_A, h_A, sizeof(hipFloatComplex)*N, hipMemcpyHostToDevice));

    for(int i = 0; i < 10; ++i) printf("%lf + i%lf ", hipCrealf(h_A[i]), hipCimagf(h_A[i]));
    printf("\n");

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;    
    transComplexFloat<<<blocksPerGrid, threadsPerBlock>>>(d_A, N);

    checkCuda(hipMemcpy(h_B, d_A, sizeof(hipFloatComplex)*(N), hipMemcpyDeviceToHost));  

    for(int i = 0; i < 10; ++i) printf("%f + i%f ", hipCrealf(h_B[i]), hipCimagf(h_B[i]));
    printf("\n");


  //free memmory
  hipFree(d_A);

  free(h_A);
  free(h_B);

  
  return 0;

}
