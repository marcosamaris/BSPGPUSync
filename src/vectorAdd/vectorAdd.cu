#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */

/* Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 3
 * of the programming guide with some additions like error checking.
 *
 */
//#include <prof.cu>
// Includes
#include <stdio.h>
#include <assert.h>    
#include <hip/hip_runtime_api.h> 
//#include <cutil_inline.h>

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

// Variables
float* h_A;
float* h_B;
float* h_C;
float* d_A;
float* d_B;
float* d_C;
bool noprompt = false;

// Functions
void Cleanup(void);
void RandomInit(float*, int);
void ParseArguments(int, char**);

// Device code
__global__ void vectorAdd(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

// Host code
int main(int argc, char** argv)
{

    if (argc != 3 ) {
		fprintf(stderr, "Syntax: %s <Vector size>  <device>\n", argv[0]);
    		return EXIT_FAILURE;
	}
    
    int N = atoi(argv[1]);
    int devId = atoi(argv[2]);

    size_t size = N * sizeof(float);
    
    checkCuda( hipSetDevice(devId) );
    hipDeviceReset();
    
    ParseArguments(argc, argv);

    printf("Vector addition\n");
    

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    if (h_A == 0) Cleanup();
    h_B = (float*)malloc(size);
    if (h_B == 0) Cleanup();
    h_C = (float*)malloc(size);
    if (h_C == 0) Cleanup();

    // Initialize input vectors
    RandomInit(h_A, N);
    RandomInit(h_B, N);

    // Allocate vectors in device memory
    checkCuda(hipMalloc((void**)&d_A, size)) ;
    checkCuda(hipMalloc((void**)&d_B, size)) ;
    checkCuda(hipMalloc((void**)&d_C, size)) ;

    // Copy vectors from host memory to device memory
    checkCuda(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice)) ;
    checkCuda(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice)) ;



    // Invoke kernel
    int threadsPerBlock = Tile_Width*Tile_Width;
    int GridSize = (N + threadsPerBlock - 1) / threadsPerBlock;
//  GpuProfiling::prepareProfiling( blocksPerGrid, threadsPerBlock );
    vectorAdd<<<GridSize, threadsPerBlock>>>(d_A, d_B, d_C, N);
//  GpuProfiling::addResults("VecAdd");
//    cutilCheckMsg("kernel launch failure");
#ifdef _DEBUG
    cutilSafeCall( hipDeviceSynchronize() );
#endif

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    checkCuda(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost)) ;

    // Verify result
    int i;
    for (i = 0; i < N; ++i) {
        float sum = h_A[i] + h_B[i];
        if (fabs(h_C[i] - sum) > 1e-5)
            break;
    }
    printf("Assertion started\n");
    printf("%s \n", (i == N) ? "PASSED" : "FAILED");    
    assert(i == N);
    printf("Assertion Finished");

    Cleanup();
    return 0;
}

void Cleanup(void)
{
    // Free device memory
    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);

    hipDeviceReset() ;
exit(0);
    if (!noprompt) {
        printf("\nPress ENTER to exit...\n");
        fflush( stdout);
        fflush( stderr);
        getchar();
    }

    exit(0);
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

// Parse program arguments
void ParseArguments(int argc, char** argv)
{
    for (int i = 0; i < argc; ++i)
        if (strcmp(argv[i], "--noprompt") == 0 ||
            strcmp(argv[i], "-noprompt") == 0)
        {
            noprompt = true;
            break;
        }
}
