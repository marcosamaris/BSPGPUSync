#include <complex>
#include <stdio.h>
#include <cmath>
#include <float.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h>


inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

hipDoubleComplex *h_A;
hipDoubleComplex *h_B;
hipDoubleComplex z;

__global__ void transComplexDouble(hipDoubleComplex *array, int N){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        i++;
}

int main(int argc, char** argv ) {
    
    if (argc != 3 ) {
        fprintf(stderr, "Syntax: %s <Vector size>  <device>\n", argv[0]);
            return EXIT_FAILURE;
    }
    hipProfilerStart();
    
    int N = atoi(argv[1]);    
    size_t size = N * sizeof(double);
    
    int devId = atoi(argv[2]);
    checkCuda( hipSetDevice(devId) );
    hipDeviceReset();

    hipDeviceProp_t prop;
    checkCuda( hipGetDeviceProperties(&prop, devId) );
    printf("Device: %s\n", prop.name);

    h_A = new hipDoubleComplex[N];
    h_B = new hipDoubleComplex[N];

    for(int i = 0; i < N; ++i){
        double Ti = ((rand() / (float)RAND_MAX)*DBL_MAX) + (rand() / (float)RAND_MAX);
        double Tj = ((rand() / (float)RAND_MAX)*DBL_MAX) + (rand() / (float)RAND_MAX);
        z = make_hipDoubleComplex(Ti, Tj);
        h_A[i] = make_hipDoubleComplex(hipCreal(z), hipCimag(z));
        h_B[i] = make_hipDoubleComplex(0., 0.);      
    }

  
  //Allocate and copy memory to device
  hipDoubleComplex *d_A;
  checkCuda(hipMalloc((void**)&d_A, sizeof(hipDoubleComplex)*N));
  checkCuda(hipMemcpy(d_A, h_A, sizeof(hipDoubleComplex)*N, hipMemcpyHostToDevice));

    for(int i = 0; i < 10; ++i) printf("%lf + i%lf ", hipCreal(h_A[i]), hipCimag(h_A[i]));
    printf("\n");

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;    
    transComplexDouble<<<blocksPerGrid, threadsPerBlock>>>(d_A, N);

    checkCuda(hipMemcpy(h_B, d_A, sizeof(hipDoubleComplex)*(N), hipMemcpyDeviceToHost));  

    for(int i = 0; i < 10; ++i) printf("%lf + i%lf ", hipCreal(h_B[i]), hipCimag(h_B[i]));
    printf("\n");


  //free memmory
  hipFree(d_A);

  free(h_A);
  free(h_B);

  
  return 0;

}
