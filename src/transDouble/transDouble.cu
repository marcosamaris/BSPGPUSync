#include "hip/hip_runtime.h"
//#include <prof.cu>
// Includes
#include <stdio.h>
#include <float.h>
#include <assert.h>     
#include <hip/hip_runtime_api.h>
//#include <cutil_inline.h>

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}


// Variables
double* h_A;
double* h_B;
double* d_A;

// Functions
void RandomInit(double*, int);

// Device code
__global__ void transDouble(double* A, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        i++;
}

// Host code
int main(int argc, char** argv)
{

    if (argc != 3 ) {
        fprintf(stderr, "Syntax: %s <Vector size>  <device>\n", argv[0]);
            return EXIT_FAILURE;
    }
    hipProfilerStart();
    
    int N = atoi(argv[1]);    
    size_t size = N * sizeof(double);
    
    int devId = atoi(argv[2]);
    checkCuda( hipSetDevice(devId) );
    hipDeviceReset();

    hipDeviceProp_t prop;
    checkCuda( hipGetDeviceProperties(&prop, devId) );
    printf("Device: %s\n", prop.name);

    // Allocate input vectors h_A and h_B in host memory
    h_A = (double*)malloc(size);
    h_B = (double*)malloc(size);

    // Initialize input vectors
    RandomInit(h_A, N);

    // Allocate vectors in device memory
    checkCuda(hipMalloc((void**)&d_A, size * sizeof(double))) ;

    // Copy vectors from host memory to device memory
    checkCuda(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice)) ;

    for(int i; i<10; i++) printf("%lf, ", h_A[i]);
    printf("\n");

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
//  GpuProfiling::prepareProfiling( blocksPerGrid, threadsPerBlock );
    transDouble<<<blocksPerGrid, threadsPerBlock>>>(d_A, N);
//  GpuProfiling::addResults("VecAdd");
//    cutilCheckMsg("kernel launch failure");
#ifdef _DEBUG
    checkCuda( hipDeviceSynchronize() );
#endif

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    checkCuda(hipMemcpy(h_B, d_A, size, hipMemcpyDeviceToHost)) ;

    for(int i; i<10; i++) printf("%lf, ", h_B[i]);
    printf("\n");

    free(h_A);
    free(h_B);
    hipFree(d_A);
    hipDeviceReset() ;

    return 0;
    hipProfilerStop();

}



// Allocates an array with random double entries.
void RandomInit(double* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = ((rand() / (float)RAND_MAX)*DBL_MAX) + (rand() / (float)RAND_MAX);
}

